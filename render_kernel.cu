#include "hip/hip_runtime.h"
/**
 * Ian Sohl & Xin Tong - 2015
 * Copyright (c) 2015      Los Alamos National Security, LLC
 *                         All rights reserved.
 * Legion Image Composition - Ray Trace Rendering Code
 */

#ifndef RENDER_CU
#define RENDER_CU

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
// #include "hip/hip_runtime_api.h"
#include "cuda_helper.h"


#include "composite.h"
#include "hip/hip_vector_types.h"
#include "CUDAMarchingCubes.h"

// #include "QMatrix4x4"


typedef float VolumeType;


typedef struct
{
	float4 m[4];
} float4x4; /**< Matrix Holding form */

__device__
float4 mul(const float4x4 &M, const float4 &v){
	/**
	 * Multiply a 4x4 Matrix with a 1x4 vector
	 */
	float4 r;
	r.w = dot(v, M.m[3]);
	r.x = dot(v, M.m[0]);
	r.y = dot(v, M.m[1]);
	r.z = dot(v, M.m[2]);

	return r;
}

__device__
float4 divW(float4 v){
	/**
	 * Divide a 4-vector by it's homogeneous coordinate
	 */
	float invW = 1 / v.w;
	return(make_float4(v.x * invW, v.y * invW, v.z * invW, 1.0f));
}



struct MyRay{
	float3 o;   /**< Origin Point */
	float3 d;   /**< Direction Vector */
}; /**< Ray vector representation */



__device__
int intersectBox(MyRay r, float3 boxmin, float3 boxmax, float *tnear, float *tfar){
	/**
	 * Check if a ray intersects with the data partition
	 */
	// compute intersection of ray with all six bbox planes
	float3 invR = make_float3(1.0f) / r.d;
	float3 tbot = invR * (boxmin - r.o);
	float3 ttop = invR * (boxmax - r.o);

	// re-order intersections to find smallest and largest on each axis
	float3 tmin = fminf(ttop, tbot);
	float3 tmax = fmaxf(ttop, tbot);

	// find the largest tmin and the smallest tmax
	float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
	float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

	*tnear = largest_tmin;
	*tfar = smallest_tmax;

	return smallest_tmax > largest_tmin;
}


__device__
void drawPixel(float* imgPtr, int x, int y, int imageW, float r, float g, float b, float a){
	/**
	 * Populate a Legion region with a particular pixel
	 */
	int writePoint = (y*imageW+x)*4; // X->Y ordering
	imgPtr[writePoint++] = r;
	imgPtr[writePoint++] = g;
	imgPtr[writePoint++] = b;
	imgPtr[writePoint] = a;
}

__device__
float interpolate(float* dataPtr, float3 pos, int3 partitionStart, int3 partitionSize){
	/**
	 * Replicate Texture functionality with a trilinear interpolant
	 */
	int3 originPoint = make_int3(floor(pos.x),floor(pos.y),floor(pos.z)); 		// Find the corner of the box the point is in
	float3 point = pos-make_float3(originPoint.x,originPoint.y,originPoint.z);	// Find the location of the point within the box
	float3 complement = make_float3(1,1,1)-point;								// Compute the distance to the opposite corner
	auto getPoint = [&](int x, int y, int z){									// Lambda function: Get a particular point from volumetric data
	int3 p = originPoint + make_int3(x,y,z);									//		Only works on integer values
		if(p.x>=partitionStart.x+partitionSize.x || p.x<partitionStart.x ||		// 	Make sure the point is in the array
				p.y>=partitionStart.y+partitionSize.y || p.y<partitionStart.y || 
				p.z>=partitionStart.z+partitionSize.z || p.z<partitionStart.z)
			return 0.0f;
		else
			return dataPtr[p.z*partitionSize.y*partitionSize.x+p.y*partitionSize.x+p.x];	// Get the point from legion X->Y->Z
	};
	float sample = 	getPoint(0,0,0) *	complement.x *	complement.y *	complement.z +	// Standard trilinear interpolant
					getPoint(1,0,0) *	point.x		 *	complement.y *	complement.z +
					getPoint(0,1,0) *	complement.x *	point.y		 *	complement.z +
					getPoint(0,0,1) *	complement.x *	complement.y *	point.z 	 +
					getPoint(1,0,1) *	point.x		 *	complement.y *	point.z		 +
					getPoint(0,1,1) *	complement.x *	point.y		 *	point.z		 +
					getPoint(1,1,0) *	point.x		 *	point.y		 *	complement.z +
					getPoint(1,1,1) *	point.x		 *	point.y		 *	point.z;
	return sample;
}


__global__ void
d_render(int imageW, int imageH,
		int3 boxStart, int3 boxSize,
		int3 partitionStart, int3 partitionSize,
		float density, float brightness,
		float transferOffset, float transferScale, 
		float* imgPtr,
		float4x4 invPVMMatrix, float* dataPtr)
{
	/**
	 * Kernal renderer for individual ray tracing
	 */

	const int maxSteps = (int)sqrtf(partitionSize.x*partitionSize.x+partitionSize.y*partitionSize.y+partitionSize.z*partitionSize.z);	// The maximum possible number of steps
	const float tstep = 1.0f;				// Distance to step
	const float opacityThreshold = 0.95f;	// Arbitrarily defined alpha cutoff

	const float3 boxMin = make_float3(boxStart.x, boxStart.y, boxStart.z); 	// Minimum bounds of data partition
	const float3 boxMax = make_float3(boxMin.x + boxSize.x - 1,				// Maximum bound of partition
			boxMin.y + boxSize.y - 1,
			boxMin.z + boxSize.z - 1);

	uint x = blockIdx.x*blockDim.x + threadIdx.x;	// Current pixel x value
	uint y = blockIdx.y*blockDim.y + threadIdx.y;	// Current pixel y value
	

	if ((x >= imageW) || (y >= imageH)) return;


	drawPixel(imgPtr,x,y,imageW,0.0f,0.0f,0.0f,0.0f);	// Fill pixel with blank

	float u = (x / (float)imageW)*2.0f-1.0f;			// Get the image space coordinates
	float v = (y / (float)imageH)*2.0f-1.0f;

	//unproject eye ray from clip space to object space
	//unproject: http://gamedev.stackexchange.com/questions/8974/how-can-i-convert-a-mouse-click-to-a-ray
	MyRay eyeRay;
	eyeRay.o = make_float3(divW(mul(invPVMMatrix, make_float4(u, v, 2.0f, 1.0f))));
	float3 eyeRay_t = make_float3(divW(mul(invPVMMatrix, make_float4(u, v, -1.0f, 1.0f))));
	eyeRay.d = normalize(eyeRay_t - eyeRay.o);

	// find intersection with box
	float tnear, tfar;
	int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);
	float4 cols[] = { 	// Hard-coded transfer function (Fixme)
			make_float4(0.0, 0.0, 0.0, 0.0),
			make_float4(1.0, 0.0, 0.0, 1.0),
			make_float4(1.0, 0.5, 0.0, 1.0),
			make_float4(1.0, 1.0, 0.0, 1.0),
			make_float4(0.0, 1.0, 0.0, 1.0),
			make_float4(0.0, 1.0, 1.0, 1.0),
			make_float4(0.0, 0.0, 1.0, 1.0),
			make_float4(1.0, 0.0, 1.0, 1.0)
	};

	if (hit){
		// drawPixel(imgPtr,x,y,imageW,(float)1,(float)1,(float)0,(float)1);
		// return;

		if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

		// march along ray from front to back, accumulating color
		float4 sum = make_float4(0.0f,0.0f,0.0f,0.0f);
		float t = tnear;
		float3 pos = eyeRay.o + eyeRay.d*tnear;
		float3 step = eyeRay.d*tstep;

		for (int i=0; i<maxSteps; i++){
			if(pos.x< boxMax.x && pos.x >= boxMin.x && pos.y< boxMax.y && pos.y >= boxMin.y && pos.z< boxMax.z && pos.z >= boxMin.z){

//				float sample = tex3D<float>(tex, pos.x, pos.y , pos.z);//interpolate(dataPtr,pos,partitionStart,partitionSize);//
				float sample = interpolate(dataPtr,pos,partitionStart,partitionSize);
				// lookup in transfer function texture
				//		    float4 col = tex1D(transferTex, (sample-transferOffset)*transferScale);
				float4 col;
//				if(sample>=8.0/36.0 || sample<0){
//					col=make_float4(0,0,0,0);
//				}
//				else{
					col = cols[(int)floor(sample*8)];
//				}
				col.w *= density;

				// "under" operator for back-to-front blending
				//sum = lerp(sum, col, col.w);

				// pre-multiply alpha
				col.x *= col.w;
				col.y *= col.w;
				col.z *= col.w;
				// "over" operator for front-to-back blending
				sum += col*(1.0f - sum.w);

				// exit early if opaque
				if (sum.w > opacityThreshold)
					break;
				
			}

			t += tstep;
			if (t > tfar) break;
			pos += step;
		}

		sum *= brightness;
		
		drawPixel(imgPtr,x,y,imageW,(float)sum.x,(float)sum.y,(float)sum.z,(float)sum.w);
	}
}


__host__
int iDivUp(int a, int b){
	/**
	 * Integer division with rounding up
	 */
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

template<typename T>
inline T* GetDeviceArray(int nv)
{
    T* array;
    if(hipMalloc((void**) &array, sizeof(T) * nv) != hipSuccess) {
        std::cout<<"memory allocation failed..."<<std::endl;
        exit(2);
    }
    return array;
}

#ifdef ISOSURFACE
__host__
void create_isosurface_task(const Task *task,
		const std::vector<PhysicalRegion> &regions,
		LegionRuntime::HighLevel::Context ctx, HighLevelRuntime *runtime){
	compositeArguments co = *((compositeArguments*)task->args);

	PhysicalRegion metadataPhysicalRegion = regions[0];
	LogicalRegion metadataLogicalRegion = metadataPhysicalRegion.get_logical_region();
	IndexSpace metadataIndexSpace = metadataLogicalRegion.get_index_space();
	Domain totalDomain = runtime->get_index_space_domain(ctx,metadataIndexSpace);
	Rect<1> totalRect = totalDomain.get_rect<1>();	// Get metadata value index


	RegionAccessor<AccessorType::Generic, Image> filenameAccessor = regions[0].get_field_accessor(FID_META).typeify<Image>();
	Image tmpimg = filenameAccessor.read(DomainPoint::from_point<1>(Point<1>(totalRect.lo.x[0])));	// Metadata for current render

	RegionAccessor<AccessorType::Generic, float> dataAccessor = regions[2].get_field_accessor(FID_VAL).typeify<float>(); // Accessor for data


	Domain dataDomain = runtime->get_index_space_domain(ctx,regions[2].get_logical_region().get_index_space());
	Rect<1> dataRect = dataDomain.get_rect<1>();	// Get data size domain
	Rect<1> dataSubRect;							// Empty filler rectangle
	ByteOffset dataOffsets[1];						// Byte Offset object
	float* dataPtr = dataAccessor.raw_rect_ptr<1>(dataRect,dataSubRect,dataOffsets); // Get raw framebuffer pointers
		
	
	int nx = 216; int ny = 320; int nz = 320;	
	//for the use of Kepler Texture Objects, refer these two links:
	//http://devblogs.nvidia.com/parallelforall/cuda-pro-tip-kepler-texture-objects-improve-performance-and-flexibility/
	//http://stackoverflow.com/questions/24981310/cuda-create-3d-texture-and-cudaarray3d-from-device-memory
	//hipArray Descriptor
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	//cuda Array
	hipArray *d_cuArr;
	checkCudaErrors(hipMalloc3DArray(&d_cuArr, &channelDesc, make_hipExtent(nx, ny, nz), 0));
	hipMemcpy3DParms copyParams = {0};

    //Array creation
    copyParams.srcPtr   = make_hipPitchedPtr(dataPtr, nx*sizeof(float), ny, nz);
    copyParams.dstArray = d_cuArr;
    copyParams.extent   = make_hipExtent(nx,ny,nz);
    copyParams.kind     = hipMemcpyDeviceToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));
    //Array creation End

	// create texture object
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = d_cuArr;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.normalizedCoords = true;
	texDesc.filterMode = hipFilterModeLinear;	//this means trilinear interpolation
	texDesc.addressMode[0] = hipAddressModeBorder;   // border: outside is 0
	texDesc.addressMode[1] = hipAddressModeBorder;
	texDesc.addressMode[2] = hipAddressModeBorder;
	texDesc.readMode = hipReadModeElementType;
	// create texture object: we only have to do this once!
	hipTextureObject_t tex=0;
	hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
	
	
	/***Computing Isosurface (Marching Cubes)***/
	CUDAMarchingCubes* mc = new CUDAMarchingCubes();
	mc->Initialize(make_uint3(nx, ny, nz));
	float3* null_pointer = NULL;

	
	int totalVerts = 0;
	float isovalue = 0.03;
	mc->SetIsovalue(isovalue);

	mc->SetVolumeData(dataPtr, tex,
			null_pointer,
			make_uint3(nx, ny, nz), make_float3(0,0,0), make_float3(nx, ny, nz), true);


	//TODO: this boundary needs to be fixed in the real simulation
	uint3 volstart = make_uint3(tmpimg.partition.xmin, tmpimg.partition.ymin, tmpimg.partition.zmin);
	uint3 volend = make_uint3(tmpimg.partition.xmax - 1,tmpimg.partition.ymax - 1,tmpimg.partition.zmax - 1);
	uint3 slabsz = volend - volstart + make_uint3(1,1,1);
	printf("volstart: %d, %d, %d  \n", volstart.x, volstart.y, volstart.z);
	printf("volend: %d, %d, %d  \n", volend.x, volend.y, volend.z);
	mc->SetSubVolume(volstart, volend);

	Domain surfaceDomain = runtime->get_index_space_domain(ctx,regions[1].get_logical_region().get_index_space());
	Rect<1> surfaceRect = surfaceDomain.get_rect<1>();
	Rect<1> surfaceSubRect;
	ByteOffset surfaceOffsets[1];
	
	RegionAccessor<AccessorType::Generic, float3> VertexAccessor = regions[1].get_field_accessor(FID_VERTEX).typeify<float3>();
	float3* VertexPtr = VertexAccessor.raw_rect_ptr<1>(surfaceRect,surfaceSubRect,surfaceOffsets);
	RegionAccessor<AccessorType::Generic, float3> NormalAccessor = regions[1].get_field_accessor(FID_NORMAL).typeify<float3>();
	float3* NormalPtr = NormalAccessor.raw_rect_ptr<1>(surfaceRect,surfaceSubRect,surfaceOffsets);

	int chunkmaxverts = slabsz.x * slabsz.y * slabsz.z;
	// float3* v3f_chunk_d = GetDeviceArray<float3>(chunkmaxverts);
	// float3* n3f_chunk_d = GetDeviceArray<float3>(chunkmaxverts);
	mc->computeIsosurface(VertexPtr, NormalPtr, (float3*)0, chunkmaxverts);
	
	int chunknumverts = mc->GetVertexCount();
	float chunkArea = mc->computeSurfaceArea(VertexPtr, chunknumverts / 3);
	printf("number of vertex: %d, surface area: %f \n", chunknumverts, chunkArea);
	
	Domain ntriSurfaceDomain = runtime->get_index_space_domain(ctx,regions[3].get_logical_region().get_index_space());
	Rect<1> ntriSurfaceRect = ntriSurfaceDomain.get_rect<1>();
	Rect<1> ntriSurfaceSubRect;
	ByteOffset ntriSurfaceOffsets[1];
	RegionAccessor<AccessorType::Generic, int> ntriAccessor = regions[3].get_field_accessor(FID_NTRI).typeify<int>();
	ntriAccessor.write(DomainPoint::from_point<1>(Point<1>(0)),chunknumverts/3);
	// int* ntriPtr = ntriAccessor.raw_rect_ptr<1>(ntriSurfaceRect,ntriSurfaceSubRect,ntriSurfaceOffsets);
	// *ntriPtr = chunknumverts / 3;
	
	// //CHECKME: I have no clue if this is the proper way to iterate
	// assert(chunkmaxverts==surfaceRect.volume());
	// for(int i = 0; i < chunkmaxverts; ++i){
	// 	VertexPtr[i] = v3f_chunk_d[i];
	// 	NormalPtr[i] = n3f_chunk_d[i];
	// }
	

	

	

	// hipFree(v3f_chunk_d);
	// hipFree(n3f_chunk_d);

	/********End of Marching Cubes*********/
	hipDestroyTextureObject(tex);
}

#endif

__host__
void create_task(const Task *task,
		const std::vector<PhysicalRegion> &regions,
		LegionRuntime::HighLevel::Context ctx, HighLevelRuntime *runtime){
	/**
	 * Image rendering task
	 */
	printf("Started Render\n");
	
	assert(regions.size()==3);
	compositeArguments co = *((compositeArguments*)task->args);

	PhysicalRegion metadataPhysicalRegion = regions[0];
	LogicalRegion metadataLogicalRegion = metadataPhysicalRegion.get_logical_region();
	IndexSpace metadataIndexSpace = metadataLogicalRegion.get_index_space();
	Domain totalDomain = runtime->get_index_space_domain(ctx,metadataIndexSpace);
	Rect<1> totalRect = totalDomain.get_rect<1>();	// Get metadata value index

//	printf("1");
//	printf("a");
	RegionAccessor<AccessorType::Generic, Image> filenameAccessor = regions[0].get_field_accessor(FID_META).typeify<Image>();
//	printf("b");
	Image tmpimg = filenameAccessor.read(DomainPoint::from_point<1>(Point<1>(totalRect.lo.x[0])));	// Metadata for current render
//	printf("c");
//	printf("d");
	RegionAccessor<AccessorType::Generic, float> dataAccessor = regions[2].get_field_accessor(FID_VAL).typeify<float>(); // Accessor for data
//	printf("e");
	RegionAccessor<AccessorType::Generic, float> imgAccessor = regions[1].get_field_accessor(FID_VAL).typeify<float>();	// And image
//	printf("f");
	float density = 0.05f;			// Arbitrary defined constants
	float brightness = 1.0f;		// 	(should be moved into metadata)
	float transferOffset = 0.0f;
	float transferScale = 1.0f;
	int width = co.width;			// Get total image size
	int height = co.height;

//	printf("g");

	float4x4 invPVMMatrix; // Copy over inverse PV Matrix from metadata
	for(int i = 0; i < 4; ++i){
		invPVMMatrix.m[i].x = tmpimg.invPVM[4*i+0];
		invPVMMatrix.m[i].y = tmpimg.invPVM[4*i+1];
		invPVMMatrix.m[i].z = tmpimg.invPVM[4*i+2];
		invPVMMatrix.m[i].w = tmpimg.invPVM[4*i+3];
	}
//	printf("h");

	dim3 blockSize = dim3(16,16);	// Define kernal execution block size
	dim3 gridSize = dim3(iDivUp(width, blockSize.x), iDivUp(height, blockSize.y)); // Number of pixels per block

//	printf("2");

	Domain dataDomain = runtime->get_index_space_domain(ctx,regions[2].get_logical_region().get_index_space());
	Rect<1> dataRect = dataDomain.get_rect<1>();	// Get data size domain
	Rect<1> dataSubRect;							// Empty filler rectangle
	ByteOffset dataOffsets[1];						// Byte Offset object
	float* dataPtr = dataAccessor.raw_rect_ptr<1>(dataRect,dataSubRect,dataOffsets); // Get raw framebuffer pointers
	

	Domain imgDomain = runtime->get_index_space_domain(ctx,regions[1].get_logical_region().get_index_space());
	Rect<1> imgRect = imgDomain.get_rect<1>();
	Rect<1> imgSubRect;
	ByteOffset imgOffsets[1];
	float* imgPtr = imgAccessor.raw_rect_ptr<1>(imgRect,imgSubRect,imgOffsets);	// For output image as well

	int3 lowerBound = make_int3(tmpimg.partition.xmin, tmpimg.partition.ymin, tmpimg.partition.zmin);
	int3 upperBound = make_int3(tmpimg.partition.xmax,tmpimg.partition.ymax,tmpimg.partition.zmax);

//	printf("3");

	int nx = 512; int ny = 512; int nz = 182;

	//for the use of Kepler Texture Objects, refer these two links:
	//http://devblogs.nvidia.com/parallelforall/cuda-pro-tip-kepler-texture-objects-improve-performance-and-flexibility/
	//http://stackoverflow.com/questions/24981310/cuda-create-3d-texture-and-cudaarray3d-from-device-memory
	//hipArray Descriptor
	//hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	//cuda Array
//	hipArray *d_cuArr;
	//checkCudaErrors(hipMalloc3DArray(&d_cuArr, &channelDesc, make_hipExtent(nx, ny, nz), 0));
//	hipMemcpy3DParms copyParams = {0};

//	printf("4");

    //Array creation
//    copyParams.srcPtr   = make_hipPitchedPtr(dataPtr, nx*sizeof(float), ny, nz);
//    copyParams.dstArray = d_cuArr;
//    copyParams.extent   = make_hipExtent(nx,ny,nz);
//    copyParams.kind     = hipMemcpyDeviceToDevice;
    //checkCudaErrors(hipMemcpy3D(&copyParams));
    //Array creation End

	// create texture object
//	hipResourceDesc resDesc;
//	memset(&resDesc, 0, sizeof(resDesc));
//	resDesc.resType = hipResourceTypeArray;
//	resDesc.res.array.array = d_cuArr;

//	printf("5");

//	hipTextureDesc texDesc;
//	memset(&texDesc, 0, sizeof(texDesc));
//	texDesc.normalizedCoords = false;
//	texDesc.filterMode = hipFilterModeLinear;	//this means trilinear interpolation
//	texDesc.addressMode[0] = hipAddressModeBorder;   // border: outside is 0
//	texDesc.addressMode[1] = hipAddressModeBorder;
//	texDesc.addressMode[2] = hipAddressModeBorder;
//	texDesc.readMode = hipReadModeElementType;
	// create texture object: we only have to do this once!
//	hipTextureObject_t tex=0;
	//hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
//	printf("6");
	d_render<<<gridSize, blockSize>>>(width,height,lowerBound,upperBound - lowerBound + make_int3(1,1,1),make_int3(0,0,0),make_int3(nx,ny,nz),density,brightness,transferOffset,transferScale,imgPtr,invPVMMatrix, dataPtr);
//	d_render<<<gridSize, blockSize>>>(width,height,make_int3(0,0,0),make_int3(nx,ny,nz),make_int3(0,0,0),make_int3(nx,ny,nz),density,brightness,transferOffset,transferScale,imgPtr,invPVMMatrix, dataPtr);
//	printf("7");
	//hipDestroyTextureObject(tex);

	//this following line causes the program to crash for unknown reason
	//hipFree(d_cuArr);

	hipDeviceSynchronize();
//	printf("CUDA: %d",(int)tmpimg.order);

}
#endif
